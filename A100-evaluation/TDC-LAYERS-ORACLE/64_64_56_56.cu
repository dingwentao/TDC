#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>
#define TH 1
#define TW 8
#define TC 16
#define C 64
#define N 64
#define H 56
#define W 56

#define TCS ((C-1)/TC + 1)
#define THS ((H-1)/TH + 1)
#define TWS ((W-1)/TW+1)
#define WPAD (TWS*TW + 2)
#define R 3
#define S 3


using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[16];
  __shared__ float pad_temp_shared[2784];
  __shared__ float kernel_shared[1152];
  float pad_temp_shared_local[32];
  float kernel_shared_local[32];
  #pragma unroll
  for (int ff_c_init = 0; ff_c_init < 2; ++ff_c_init) {
    compute_local[(ff_c_init)] = 0.000000e+00f;
    compute_local[((ff_c_init + 8))] = 0.000000e+00f;
    compute_local[((ff_c_init + 2))] = 0.000000e+00f;
    compute_local[((ff_c_init + 10))] = 0.000000e+00f;
    compute_local[((ff_c_init + 4))] = 0.000000e+00f;
    compute_local[((ff_c_init + 12))] = 0.000000e+00f;
    compute_local[((ff_c_init + 6))] = 0.000000e+00f;
    compute_local[((ff_c_init + 14))] = 0.000000e+00f;
  }
  for (int rc_outer = 0; rc_outer < 8; ++rc_outer) {
    __syncthreads();
    #pragma unroll
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 13; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
      if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.y) * 13) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 348)) < 8) {
        if (((((int)threadIdx.z) * 116) + (((((int)threadIdx.y) * 13) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 6)) < 464) {
          if ((((((int)threadIdx.z) * 696) + (((int)threadIdx.y) * 13)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 2784) {
            if (((((int)threadIdx.y) * 13) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 696) {
              pad_temp_shared[((((((int)threadIdx.z) * 696) + (((int)threadIdx.y) * 13)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = (((((6 <= (((((int)threadIdx.y) * 13) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 348)) && ((((((int)threadIdx.y) * 13) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 348) < 342)) && (1 <= ((((int)blockIdx.x) * 4) + (((((int)threadIdx.y) * 13) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 6)))) && (((((int)blockIdx.x) * 4) + (((((int)threadIdx.y) * 13) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 6)) < 57)) ? data[((((((((rc_outer * 25088) + (((int)threadIdx.z) * 6272)) + ((((((int)threadIdx.y) * 13) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 348) * 3136)) + (((((((int)threadIdx.y) * 13) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 348) / 6) * 56)) + (((int)blockIdx.x) * 4)) + (((((int)threadIdx.y) * 13) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 6)) - 57))] : 0.000000e+00f);
            }
          }
        }
      }
    }
    #pragma unroll
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 < 6; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) {
      if (((((int)threadIdx.z) * 4) + (((((int)threadIdx.y) * 2) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 / 3)) / 24)) < 16) {
        if (((((int)threadIdx.z) * 32) + (((((int)threadIdx.y) * 2) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 / 3)) / 3)) < 128) {
          if ((((((int)threadIdx.z) * 96) + (((int)threadIdx.y) * 2)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 / 3)) < 384) {
            if ((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 6)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 1152) {
              if (((((int)threadIdx.y) * 6) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 288) {
                if ((((((int)blockIdx.z) * 16) + (((int)threadIdx.z) * 4)) + (((((int)threadIdx.y) * 2) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 / 3)) / 24)) < 64) {
                  kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 6)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))] = kernel[(((((((((int)blockIdx.z) * 9216) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.y) * 2) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 / 3)) / 24) * 576)) + (rc_outer * 72)) + ((((((int)threadIdx.y) * 2) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 / 3)) % 24) * 3)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 % 3)))];
                }
              }
            }
          }
        }
      }
    }
    __syncthreads();
    for (int ry_inner_outer = 0; ry_inner_outer < 3; ++ry_inner_outer) {
      #pragma unroll
      for (int rx_inner_outer = 0; rx_inner_outer < 3; ++rx_inner_outer) {
        #pragma unroll
        for (int ax1 = 0; ax1 < 8; ++ax1) {
          pad_temp_shared_local[(ax1)] = pad_temp_shared[(((((ax1 * 348) + (((int)threadIdx.y) * 6)) + (ry_inner_outer * 6)) + rx_inner_outer))];
          pad_temp_shared_local[((ax1 + 8))] = pad_temp_shared[((((((ax1 * 348) + (((int)threadIdx.y) * 6)) + (ry_inner_outer * 6)) + rx_inner_outer) + 1))];
          pad_temp_shared_local[((ax1 + 16))] = pad_temp_shared[((((((ax1 * 348) + (((int)threadIdx.y) * 6)) + (ry_inner_outer * 6)) + rx_inner_outer) + 2))];
          pad_temp_shared_local[((ax1 + 24))] = pad_temp_shared[((((((ax1 * 348) + (((int)threadIdx.y) * 6)) + (ry_inner_outer * 6)) + rx_inner_outer) + 3))];
        }
        #pragma unroll
        for (int ax0 = 0; ax0 < 2; ++ax0) {
          #pragma unroll
          for (int ax11 = 0; ax11 < 8; ++ax11) {
            kernel_shared_local[(((ax0 * 8) + ax11))] = kernel_shared[((((((((int)threadIdx.z) * 144) + (ax0 * 72)) + (ax11 * 9)) + (ry_inner_outer * 3)) + rx_inner_outer))];
            kernel_shared_local[((((ax0 * 8) + ax11) + 16))] = kernel_shared[(((((((((int)threadIdx.z) * 144) + (ax0 * 72)) + (ax11 * 9)) + (ry_inner_outer * 3)) + rx_inner_outer) + 576))];
          }
        }
        #pragma unroll
        for (int rc_inner_inner = 0; rc_inner_inner < 8; ++rc_inner_inner) {
          #pragma unroll
          for (int ff_c = 0; ff_c < 2; ++ff_c) {
            compute_local[(ff_c)] = (compute_local[(ff_c)] + (pad_temp_shared_local[(rc_inner_inner)] * kernel_shared_local[(((ff_c * 8) + rc_inner_inner))]));
            compute_local[((ff_c + 8))] = (compute_local[((ff_c + 8))] + (pad_temp_shared_local[(rc_inner_inner)] * kernel_shared_local[((((ff_c * 8) + rc_inner_inner) + 16))]));
            compute_local[((ff_c + 2))] = (compute_local[((ff_c + 2))] + (pad_temp_shared_local[((rc_inner_inner + 8))] * kernel_shared_local[(((ff_c * 8) + rc_inner_inner))]));
            compute_local[((ff_c + 10))] = (compute_local[((ff_c + 10))] + (pad_temp_shared_local[((rc_inner_inner + 8))] * kernel_shared_local[((((ff_c * 8) + rc_inner_inner) + 16))]));
            compute_local[((ff_c + 4))] = (compute_local[((ff_c + 4))] + (pad_temp_shared_local[((rc_inner_inner + 16))] * kernel_shared_local[(((ff_c * 8) + rc_inner_inner))]));
            compute_local[((ff_c + 12))] = (compute_local[((ff_c + 12))] + (pad_temp_shared_local[((rc_inner_inner + 16))] * kernel_shared_local[((((ff_c * 8) + rc_inner_inner) + 16))]));
            compute_local[((ff_c + 6))] = (compute_local[((ff_c + 6))] + (pad_temp_shared_local[((rc_inner_inner + 24))] * kernel_shared_local[(((ff_c * 8) + rc_inner_inner))]));
            compute_local[((ff_c + 14))] = (compute_local[((ff_c + 14))] + (pad_temp_shared_local[((rc_inner_inner + 24))] * kernel_shared_local[((((ff_c * 8) + rc_inner_inner) + 16))]));
          }
        }
      }
    }
  }
  #pragma unroll
  for (int ff_inner_inner_inner = 0; ff_inner_inner_inner < 2; ++ff_inner_inner_inner) {
    compute[((((((((int)blockIdx.z) * 50176) + (((int)threadIdx.z) * 6272)) + (ff_inner_inner_inner * 3136)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 4)))] = compute_local[(ff_inner_inner_inner)];
    compute[(((((((((int)blockIdx.z) * 50176) + (((int)threadIdx.z) * 6272)) + (ff_inner_inner_inner * 3136)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 4)) + 25088))] = compute_local[((ff_inner_inner_inner + 8))];
    compute[(((((((((int)blockIdx.z) * 50176) + (((int)threadIdx.z) * 6272)) + (ff_inner_inner_inner * 3136)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 4)) + 1))] = compute_local[((ff_inner_inner_inner + 2))];
    compute[(((((((((int)blockIdx.z) * 50176) + (((int)threadIdx.z) * 6272)) + (ff_inner_inner_inner * 3136)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 4)) + 25089))] = compute_local[((ff_inner_inner_inner + 10))];
    compute[(((((((((int)blockIdx.z) * 50176) + (((int)threadIdx.z) * 6272)) + (ff_inner_inner_inner * 3136)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 4)) + 2))] = compute_local[((ff_inner_inner_inner + 4))];
    compute[(((((((((int)blockIdx.z) * 50176) + (((int)threadIdx.z) * 6272)) + (ff_inner_inner_inner * 3136)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 4)) + 25090))] = compute_local[((ff_inner_inner_inner + 12))];
    compute[(((((((((int)blockIdx.z) * 50176) + (((int)threadIdx.z) * 6272)) + (ff_inner_inner_inner * 3136)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 4)) + 3))] = compute_local[((ff_inner_inner_inner + 6))];
    compute[(((((((((int)blockIdx.z) * 50176) + (((int)threadIdx.z) * 6272)) + (ff_inner_inner_inner * 3136)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 4)) + 25091))] = compute_local[((ff_inner_inner_inner + 14))];
  }
}






class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}

class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
__device__ void load_input_2_shared_memory(float *input, float *shared_input, unsigned int h_start,
                                           unsigned int h_end, unsigned int h_offset, unsigned int c_start,
                                           unsigned int warp_id, unsigned int lane_id, unsigned int warp_size){
    switch(h_offset){
        case 0:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
                for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
                    unsigned int r = i/W;
                    unsigned int s = i%W;
                    shared_input[c*(TH + 2)*(WPAD) + r * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
                }
            }
            break;
        case 1:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
                for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
                    unsigned int r = i/W;
                    unsigned int s = i%W;
                    shared_input[c*(TH + 2)*(WPAD) + (1 + r) * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
                }
            }
            break;
    }
}
__device__ __forceinline__ void switch_write_back(unsigned int write_h, unsigned int write_w, unsigned int h_out_start, unsigned int w_out_start, unsigned int n, float * outputs, float * temp_result){
	switch(write_h){
		case 1: 
 		switch(write_w){
			case 1:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 1; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 2:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 2; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 3:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 3; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 4:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 4; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 5:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 5; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 6:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 6; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 7:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 7; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 8:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 8; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
		} 
		break;
	}
}
__global__ void conv2d(float * __restrict__ input,const float * __restrict__ kernel, float * __restrict__ outputs){
    extern __shared__ float shared_input[];
    const unsigned int tile_id = blockIdx.x;
    const unsigned int tc_id = tile_id / THS;
    const unsigned int th_id = tile_id % THS;
    const unsigned int tw_id = threadIdx.x / N;
    const int h_out_start = th_id * TH;
    const int w_out_start = tw_id * TW;
    const unsigned int warp_id = tw_id;
    const unsigned int lane_id = threadIdx.x % N;
    float data_array[9];
    float temp_result[TH*TW] = {0.0f};
    for(unsigned int i=threadIdx.x;i<TC*(TH+2)*WPAD;i+=blockDim.x){
        shared_input[i] = 0.0f;
    }
    unsigned int n = lane_id;
    unsigned int c_offset = tc_id * TC;
    int h_offset = (h_out_start == 0)?1:0;
    int h_padded_start = h_out_start;
    int h_padded_end = min(h_padded_start + TH + 2, H + 2);
    int h_non_padded_start = max(h_out_start - 1, 0);
    int h_non_padded_end = min(H, h_padded_end - 1);
    __syncthreads();
    load_input_2_shared_memory(input, shared_input, h_non_padded_start, h_non_padded_end, h_offset, c_offset, warp_id, lane_id, N);
    __syncthreads();
#pragma unroll
    for(unsigned int c=0;c<TC;c++){
#pragma unroll
        for(unsigned int r=0;r<R;++r){
#pragma unroll
            for(unsigned int s=0;s<S;++s){
                data_array[r*S+s] = kernel[(c + c_offset)*N*9+r*3*N+s*N+n];
            }
        }
        		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 0]*data_array[0];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 1]*data_array[0];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 1]*data_array[1];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[0];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[1];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[2];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 3]*data_array[0];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 3]*data_array[1];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 3]*data_array[2];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 4]*data_array[0];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 4]*data_array[1];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 4]*data_array[2];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 5]*data_array[0];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 5]*data_array[1];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 5]*data_array[2];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 6]*data_array[0];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 6]*data_array[1];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 6]*data_array[2];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 7]*data_array[0];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 7]*data_array[1];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 7]*data_array[2];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 8]*data_array[1];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 8]*data_array[2];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 9]*data_array[2];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 0]*data_array[3];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[3];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[4];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[3];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[4];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[5];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[3];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[4];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[5];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 4]*data_array[3];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 4]*data_array[4];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 4]*data_array[5];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 5]*data_array[3];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 5]*data_array[4];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 5]*data_array[5];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 6]*data_array[3];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 6]*data_array[4];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 6]*data_array[5];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 7]*data_array[3];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 7]*data_array[4];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 7]*data_array[5];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 8]*data_array[4];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 8]*data_array[5];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 9]*data_array[5];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 0]*data_array[6];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[6];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[7];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[6];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[7];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[8];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[6];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[7];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[8];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 4]*data_array[6];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 4]*data_array[7];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 4]*data_array[8];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 5]*data_array[6];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 5]*data_array[7];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 5]*data_array[8];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 6]*data_array[6];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 6]*data_array[7];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 6]*data_array[8];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 7]*data_array[6];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 7]*data_array[7];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 7]*data_array[8];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 8]*data_array[7];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 8]*data_array[8];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 9]*data_array[8];

    }
    switch_write_back(min(TH, H - h_out_start), min(TW, W - w_out_start), h_out_start, w_out_start, n, outputs, temp_result);
}

float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
#pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    float *matrix;
    hipMalloc(&matrix,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    hipMemset(matrix,0,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);

    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);


    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);


        dim3 grid(14,1,4);

                dim3 block(1,56,4);

    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tvm;
    hipEventElapsedTime(&time_tvm, event_start, event_stop);
    float *out_tvm = new float[N*H*W];
    hipMemcpy(out_tvm,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    hipMemset(device_out, 0, sizeof(float)*N*H*W);

    chkerr(hipFuncSetAttribute(reinterpret_cast<const void*>(conv2d),hipFuncAttributeMaxDynamicSharedMemorySize, TC*(TH+2)*(WPAD)*4));
    hipEventRecord(event_start);
    conv2d<<<TCS*THS, N * TWS, TC*(TH+2)*(WPAD)*4>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    ofstream outfile;
    char buffer[1000];
    int ret = sprintf(buffer,"%d,%d,%d,%d,%f,%f,%f,%f,%f,%f,%f,%f,%f\n",N,C,H,W,
            cudnnFFTTime,cudnnWinogradeTimeNon,cudnnGemmTime,time_tvm,time_tdc,
            cudnnFFTTime/time_tdc,cudnnWinogradeTimeNon/time_tdc,cudnnGemmTime/time_tdc,time_tvm/time_tdc);
    outfile.open("../../evaluation_outcome/A100-layers-eval-oracle.csv", std::ios_base::app);
    outfile << buffer;


    float difference = check_diff(out_tvm, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<
                                   time_tvm<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<cudnnWinogradeTimeNon/time_tdc<<","<<
                                   cudnnGemmTime/time_tdc<<","<<time_tvm/time_tdc<<","<<difference<<endl;
    return 0;
}


