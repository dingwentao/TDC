#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>

#define C 128
#define N 96
#define H 14
#define W 14

#define R 3
#define S 3
using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[2];
  __shared__ float pad_temp_shared[768];
  __shared__ float kernel_shared[864];
  float pad_temp_shared_local[6];
  float kernel_shared_local[12];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 8; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[(((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 19) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 19) >> 4)) % 3)) < 15)) && (1 <= ((((int)threadIdx.x) * 19) & 15))) && (((((int)threadIdx.x) * 19) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 19) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 19) >> 4)) % 3) * 14)) + ((((int)threadIdx.x) * 19) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 1))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 1) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 1) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 1) & 15))) && ((((((int)threadIdx.x) * 19) + 1) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 1) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 1) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 1) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 2))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 2) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 2) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 2) & 15))) && ((((((int)threadIdx.x) * 19) + 2) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 2) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 2) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 2) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 3))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 3) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 3) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 3) & 15))) && ((((((int)threadIdx.x) * 19) + 3) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 3) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 3) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 3) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 4))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 4) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 4) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 4) & 15))) && ((((((int)threadIdx.x) * 19) + 4) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 4) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 4) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 4) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 5))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 5) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 5) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 5) & 15))) && ((((((int)threadIdx.x) * 19) + 5) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 5) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 5) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 5) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 6))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 6) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 6) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 6) & 15))) && ((((((int)threadIdx.x) * 19) + 6) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 6) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 6) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 6) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 7))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 7) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 7) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 7) & 15))) && ((((((int)threadIdx.x) * 19) + 7) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 7) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 7) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 7) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 8))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 8) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 8) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 8) & 15))) && ((((((int)threadIdx.x) * 19) + 8) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 8) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 8) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 8) & 15)) - 15))] : 0.000000e+00f);
    if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 9) >> 4)) < 48) {
      if (((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) < 759) {
        if (((int)threadIdx.x) < 13) {
          pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 9))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 9) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 9) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 9) & 15))) && ((((((int)threadIdx.x) * 19) + 9) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 9) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 9) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 9) & 15)) - 15))] : 0.000000e+00f);
        }
      }
    }
    if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 10) >> 4)) < 48) {
      if (((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) < 758) {
        if (((int)threadIdx.x) < 13) {
          pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 10))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 10) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 10) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 10) & 15))) && ((((((int)threadIdx.x) * 19) + 10) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 10) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 10) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 10) & 15)) - 15))] : 0.000000e+00f);
        }
      }
    }
    if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 11) >> 4)) < 48) {
      if (((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) < 757) {
        if (((int)threadIdx.x) < 13) {
          pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 11))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 11) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 11) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 11) & 15))) && ((((((int)threadIdx.x) * 19) + 11) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 11) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 11) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 11) & 15)) - 15))] : 0.000000e+00f);
        }
      }
    }
    if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 12) >> 4)) < 48) {
      if (((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) < 756) {
        if (((int)threadIdx.x) < 13) {
          pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 12))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 12) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 12) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 12) & 15))) && ((((((int)threadIdx.x) * 19) + 12) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 12) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 12) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 12) & 15)) - 15))] : 0.000000e+00f);
        }
      }
    }
    if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 13) >> 4)) < 48) {
      if (((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) < 755) {
        if (((int)threadIdx.x) < 13) {
          pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 13))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 13) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 13) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 13) & 15))) && ((((((int)threadIdx.x) * 19) + 13) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 13) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 13) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 13) & 15)) - 15))] : 0.000000e+00f);
        }
      }
    }
    if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 14) >> 4)) < 48) {
      if (((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) < 754) {
        if (((int)threadIdx.x) < 13) {
          pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 14))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 14) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 14) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 14) & 15))) && ((((((int)threadIdx.x) * 19) + 14) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 14) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 14) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 14) & 15)) - 15))] : 0.000000e+00f);
        }
      }
    }
    if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 15) >> 4)) < 48) {
      if (((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) < 753) {
        if (((int)threadIdx.x) < 13) {
          pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 15))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 15) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 15) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 15) & 15))) && ((((((int)threadIdx.x) * 19) + 15) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 15) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 15) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 15) & 15)) - 15))] : 0.000000e+00f);
        }
      }
    }
    if (((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 19) >> 4)) < 47) {
      if (((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) < 752) {
        if (((int)threadIdx.x) < 13) {
          pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 16))] = (((((1 <= (((int)blockIdx.y) + ((((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 19) >> 4)) + 1) % 3))) && ((((int)blockIdx.y) + ((((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 19) >> 4)) + 1) % 3)) < 15)) && (1 <= ((((int)threadIdx.x) * 19) & 15))) && (((((int)threadIdx.x) * 19) & 15) < 15)) ? data[(((((((rc_outer * 3136) + (((((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 19) >> 4)) + 1) / 3) * 196)) + (((int)blockIdx.y) * 14)) + (((((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 19) >> 4)) + 1) % 3) * 14)) + ((((int)threadIdx.x) * 19) & 15)) - 15))] : 0.000000e+00f);
        }
      }
    }
    if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 17) >> 4)) < 48) {
      if (((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) < 751) {
        if (((int)threadIdx.x) < 13) {
          pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 17))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 17) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 17) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 1) & 15))) && ((((((int)threadIdx.x) * 19) + 1) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 17) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 17) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 1) & 15)) - 15))] : 0.000000e+00f);
        }
      }
    }
    if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 18) >> 4)) < 48) {
      if (((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) < 750) {
        if (((int)threadIdx.x) < 13) {
          pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 18))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 18) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 18) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 2) & 15))) && ((((((int)threadIdx.x) * 19) + 2) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 18) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 18) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 2) & 15)) - 15))] : 0.000000e+00f);
        }
      }
    }
    kernel_shared[(((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + (((((int)threadIdx.x) * 7) / 48) * 1152)) + (rc_outer * 144)) + (((((int)threadIdx.x) * 7) % 48) * 3)))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 1))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + (((((int)threadIdx.x) * 7) / 48) * 1152)) + (rc_outer * 144)) + (((((int)threadIdx.x) * 7) % 48) * 3)) + 1))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 2))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + (((((int)threadIdx.x) * 7) / 48) * 1152)) + (rc_outer * 144)) + (((((int)threadIdx.x) * 7) % 48) * 3)) + 2))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 3))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 1) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 1) % 48) * 3)))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 4))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 1) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 1) % 48) * 3)) + 1))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 5))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 1) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 1) % 48) * 3)) + 2))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 6))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 2) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 2) % 48) * 3)))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 7))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 2) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 2) % 48) * 3)) + 1))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 8))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 2) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 2) % 48) * 3)) + 2))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 9))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 3) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 3) % 48) * 3)))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 10))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 3) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 3) % 48) * 3)) + 1))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 11))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 3) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 3) % 48) * 3)) + 2))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 12))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 4) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 4) % 48) * 3)))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 13))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 4) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 4) % 48) * 3)) + 1))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 14))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 4) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 4) % 48) * 3)) + 2))];
    if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 7) + 5) / 48)) < 6) {
      if (((((int)threadIdx.z) * 32) + (((((int)threadIdx.x) * 7) + 5) / 3)) < 96) {
        if (((((int)threadIdx.z) * 96) + (((int)threadIdx.x) * 7)) < 283) {
          if (((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) < 849) {
            if (((int)threadIdx.x) < 13) {
              kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 15))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 5) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 5) % 48) * 3)))];
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 7) + 5) / 48)) < 6) {
      if (((((int)threadIdx.z) * 32) + (((((int)threadIdx.x) * 7) + 5) / 3)) < 96) {
        if (((((int)threadIdx.z) * 96) + (((int)threadIdx.x) * 7)) < 283) {
          if (((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) < 848) {
            if (((int)threadIdx.x) < 13) {
              kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 16))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 5) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 5) % 48) * 3)) + 1))];
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 7) + 5) / 48)) < 6) {
      if (((((int)threadIdx.z) * 32) + (((((int)threadIdx.x) * 7) + 5) / 3)) < 96) {
        if (((((int)threadIdx.z) * 96) + (((int)threadIdx.x) * 7)) < 283) {
          if (((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) < 847) {
            if (((int)threadIdx.x) < 13) {
              kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 17))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 5) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 5) % 48) * 3)) + 2))];
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 7) + 6) / 48)) < 6) {
      if (((((int)threadIdx.z) * 32) + ((((int)threadIdx.x) * 7) / 3)) < 94) {
        if (((((int)threadIdx.z) * 96) + (((int)threadIdx.x) * 7)) < 282) {
          if (((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) < 846) {
            if (((int)threadIdx.x) < 13) {
              kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 18))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 6) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 6) % 48) * 3)))];
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 7) + 6) / 48)) < 6) {
      if (((((int)threadIdx.z) * 32) + ((((int)threadIdx.x) * 7) / 3)) < 94) {
        if (((((int)threadIdx.z) * 96) + (((int)threadIdx.x) * 7)) < 282) {
          if (((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) < 845) {
            if (((int)threadIdx.x) < 13) {
              kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 19))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 6) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 6) % 48) * 3)) + 1))];
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 7) + 6) / 48)) < 6) {
      if (((((int)threadIdx.z) * 32) + ((((int)threadIdx.x) * 7) / 3)) < 94) {
        if (((((int)threadIdx.z) * 96) + (((int)threadIdx.x) * 7)) < 282) {
          if (((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) < 844) {
            if (((int)threadIdx.x) < 13) {
              kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 20))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 6) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 6) % 48) * 3)) + 2))];
            }
          }
        }
      }
    }
    __syncthreads();
    for (int rc_inner_outer = 0; rc_inner_outer < 8; ++rc_inner_outer) {
      pad_temp_shared_local[(0)] = pad_temp_shared[(((rc_inner_outer * 96) + ((int)threadIdx.x)))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 1))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 2))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 48))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 49))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 50))];
      kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)))];
      kernel_shared_local[(6)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 432))];
      kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 1))];
      kernel_shared_local[(7)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 433))];
      kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 2))];
      kernel_shared_local[(8)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 434))];
      kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 9))];
      kernel_shared_local[(9)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 441))];
      kernel_shared_local[(4)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 10))];
      kernel_shared_local[(10)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 442))];
      kernel_shared_local[(5)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 11))];
      kernel_shared_local[(11)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 443))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(8)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(9)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(10)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(11)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 16))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 17))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 18))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 64))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 65))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 66))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 3))];
      kernel_shared_local[(6)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 435))];
      kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 4))];
      kernel_shared_local[(7)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 436))];
      kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 5))];
      kernel_shared_local[(8)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 437))];
      kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 12))];
      kernel_shared_local[(9)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 444))];
      kernel_shared_local[(4)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 13))];
      kernel_shared_local[(10)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 445))];
      kernel_shared_local[(5)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 14))];
      kernel_shared_local[(11)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 446))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(8)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(9)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(10)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(11)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 32))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 33))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 34))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 80))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 81))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 82))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 6))];
      kernel_shared_local[(6)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 438))];
      kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 7))];
      kernel_shared_local[(7)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 439))];
      kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 8))];
      kernel_shared_local[(8)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 440))];
      kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 15))];
      kernel_shared_local[(9)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 447))];
      kernel_shared_local[(4)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 16))];
      kernel_shared_local[(10)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 448))];
      kernel_shared_local[(5)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 17))];
      kernel_shared_local[(11)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 449))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(8)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(9)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(10)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(11)]));
    }
  }
  compute[(((((((int)blockIdx.z) * 1176) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 14)) + ((int)threadIdx.x)))] = compute_local[(0)];
  compute[((((((((int)blockIdx.z) * 1176) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 14)) + ((int)threadIdx.x)) + 588))] = compute_local[(1)];
}





float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
    #pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
void pad_input(float * x, float *y){
    #pragma omp parallel for
    for(unsigned int i=0;i<(H + 2)*(W+2)*C;++i){
        y[i] = 0.0f;
    }
    #pragma omp parallel for
    for(unsigned int c=0;c<C;++c){
        for(unsigned int h=0;h<H;++h){
            for(unsigned int w=0;w<W;++w){
                unsigned int h_padded = h + 1;
                unsigned int w_padded = w + 1;
                y[c*(H+2)*(W+2) + h_padded*(W+2) + w_padded] = x[c*(H)*(W) + h*(W) + w];
            }
        }
    }
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float * padded_input = new float[C*(H+2)*(W+2)];
    pad_input(input, padded_input);
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);


    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);

    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);

    dim3 grid(1,14,16);
    dim3 block(14,1,3);

    float * paddedInputDevice;
    chkerr(hipMalloc(&paddedInputDevice, C * (H + 2) * (W + 2) * sizeof(float)));
    chkerr(hipMemcpy(paddedInputDevice, padded_input, C * (H + 2) * (W + 2) * sizeof(float), hipMemcpyHostToDevice));
    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    float difference = check_diff(out_cudnn_host, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<cudnnWinogradeTimeNon/time_tdc<<","<<cudnnGemmTime/time_tdc<<endl;
    return 0;
}

